#include "hip/hip_runtime.h"
// -------------------------------------------------------------
// cuDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
//  $Revision:
//  $Date:
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * cr_kernel.cu
 *
 * @brief CUDPP kernel-level tridiagonal routines
 */

/** \addtogroup cudpp_kernel
  * @{
  */
/** @name tridiagonal Functions
 * @{
 */

template <class T>
__global__ void cyclic_small_systems_kernel(T *a_d, T *b_d, T *c_d, T *d_d, T *x_d)
{
    int thid = threadIdx.x;
    int blid = blockIdx.x;

    int stride = 1;

    int thid_num = blockDim.x;
    const unsigned int system_size = blockDim.x * 2;
   
    int Iteration = (int)log2(T(system_size/2));

    __syncthreads();

    extern __shared__ char shared[];

    T* a = (T*)shared;
    T* b = (T*)&a[system_size];
    T* c = (T*)&b[system_size];
    T* d = (T*)&c[system_size];
    T* x = (T*)&d[system_size];

    a[thid] = a_d[thid + blid * system_size];
    a[thid + blockDim.x] = a_d[thid + blockDim.x + blid * system_size];

    b[thid] = b_d[thid + blid * system_size];
    b[thid + blockDim.x] = b_d[thid + blockDim.x + blid * system_size];

    c[thid] = c_d[thid + blid * system_size];
    c[thid + blockDim.x] = c_d[thid + blockDim.x + blid * system_size];

    d[thid] = d_d[thid + blid * system_size];
    d[thid + blockDim.x] = d_d[thid + blockDim.x + blid * system_size];

    __syncthreads();


    //forward elimination
    for (int j = 0; j <Iteration; j++)
    {
        __syncthreads();
        stride *= 2;
        int delta = stride/2;
        if (thid < thid_num)
        {
            int i = stride * thid + stride - 1;

            if(i == system_size - 1)
            {
                T tmp = a[i] / b[i-delta];
                b[i] = b[i] - c[i-delta] * tmp;
                d[i] = d[i] - d[i-delta] * tmp;
                a[i] = -a[i-delta] * tmp;
                c[i] = 0;
            }
            else
            {
                T tmp1 = a[i] / b[i-delta];
                T tmp2 = c[i] / b[i+delta];
                b[i] = b[i] - c[i-delta] * tmp1 - a[i+delta] * tmp2;
                d[i] = d[i] - d[i-delta] * tmp1 - d[i+delta] * tmp2;
                a[i] = -a[i-delta] * tmp1;
                c[i] = -c[i+delta] * tmp2;
            }
        }
        thid_num /= 2;
    }

    if (thid < 2)
    {
      int addr1 = stride - 1;
      int addr2 = 2 * stride - 1;
      T tmp3 = b[addr2]*b[addr1]-c[addr1]*a[addr2];
      x[addr1] = (b[addr2]*d[addr1]-c[addr1]*d[addr2])/tmp3;
      x[addr2] = (d[addr2]*b[addr1]-d[addr1]*a[addr2])/tmp3;
    }

    //backward substitution
    thid_num = 2;
    for (int j = 0; j <Iteration; j++)
    {
        int delta = stride/2;
        __syncthreads();
        if (thid < thid_num)
        {
            int i = stride * thid + stride/2 - 1;
            if(i == delta - 1)
                  x[i] = (d[i] - c[i]*x[i+delta])/b[i];
            else
                  x[i] = (d[i] - a[i]*x[i-delta] - c[i]*x[i+delta])/b[i];
         }
         stride /= 2;
         thid_num *= 2;
      }

    __syncthreads();

    x_d[thid + blid * system_size] = x[thid];
    x_d[thid + blockDim.x + blid * system_size] = x[thid + blockDim.x];

}

template <class T>
void cyclic_small_systems(T *a, T *b, T *c, T *d, T *x, int system_size, int num_systems)
{
    const unsigned int num_threads_block = system_size/2;
    const unsigned int mem_size = sizeof(T)*num_systems*system_size;

    // allocate device memory input and output arrays
    T* device_a;
    T* device_b;
    T* device_c;
    T* device_d;
    T* device_x;

    CUDA_SAFE_CALL( hipMalloc( (void**) &device_a,mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &device_b,mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &device_c,mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &device_d,mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &device_x,mem_size));
 
   // copy host memory to device input array
    CUDA_SAFE_CALL( hipMemcpy( device_a, a,mem_size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( device_b, b,mem_size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( device_c, c,mem_size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( device_d, d,mem_size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( device_x, x,mem_size, hipMemcpyHostToDevice));

    // setup execution parameters
    dim3  grid(num_systems, 1, 1);
    dim3  threads(num_threads_block, 1, 1);

    cyclic_small_systems_kernel<<< grid, threads,system_size*5*sizeof(T)/*16000*/>>>(device_a, device_b, device_c, device_d, device_x);

    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy(x, device_x,mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(device_a));
    CUDA_SAFE_CALL(hipFree(device_b));
    CUDA_SAFE_CALL(hipFree(device_c));
    CUDA_SAFE_CALL(hipFree(device_d));
    CUDA_SAFE_CALL(hipFree(device_x));
}
/** @} */ // end tridiagonal functions
/** @} */ // end cudpp_kernel

